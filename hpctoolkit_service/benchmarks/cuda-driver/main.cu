#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../utils/common.h"


static size_t N = 10;


void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}


void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}


int main(int argc, char *argv[]) {

  // Init device
  hipDevice_t device;
  hipCtx_t context;
  hipModule_t module;
  hipFunction_t function;
  int device_id = 0;

  cu_init_device(device_id, device, context);
  // cu_load_module_function(module, "vecSet_1.cubin", function, "vecSet_1");
  cu_load_module_function(module, "vecSet1.cubin", function, "vecExpensive1");


  int l[N];
  hipDeviceptr_t dl;

  init(l, N);

  size_t threads = 256;
  size_t blocks = (N - 1) / threads + 1;

  DRIVER_API_CALL(hipCtxSetCurrent(context));

  DRIVER_API_CALL(hipMalloc(&dl, N * sizeof(int)));
  DRIVER_API_CALL(hipMemcpyHtoD(dl, l, N * sizeof(int)));

  void *args[4] = {
    &dl, &N
  };

  DRIVER_API_CALL(hipModuleLaunchKernel(function, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

  DRIVER_API_CALL(hipMemcpyDtoH(l, dl, N * sizeof(int)));

  DRIVER_API_CALL(hipFree(dl));


  output(l, N);

  DRIVER_API_CALL(hipCtxSynchronize());


  DRIVER_API_CALL(hipModuleUnload(module));
  DRIVER_API_CALL(hipCtxDestroy(context));
  RUNTIME_API_CALL(hipDeviceSynchronize());

  return 0;
}
